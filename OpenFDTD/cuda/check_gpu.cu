/*
check_gpu.cu

return = 0/1 : OK/NG
device : I : device number (=0,1,...)
msg    : O : GPU properties or error message
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

int check_gpu(int idevice, char msg[])
{
	hipError_t ierr;

	// check CUDA support
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount <= 0) {
		strcpy(msg, "*** There is no device supporting CUDA");
		return 1;
	}
	//printf("deviceCount=%d\n", deviceCount);

	// check device number
	if ((idevice < 0) || (idevice >= deviceCount)) {
		sprintf(msg, "*** Invalid device number = %d", idevice);
		return 1;
	}

	// set device
	ierr = hipSetDevice(idevice);
	if (ierr != hipSuccess) {
		strcpy(msg, hipGetErrorString(ierr));
		return 1;
	}

	// properties
	hipDeviceProp_t prop;
	ierr = hipGetDeviceProperties(&prop, idevice);
	if (ierr != hipSuccess) {
		sprintf(msg, hipGetErrorString(ierr));
		return 1;
	}
	if (prop.major < 3) {
		sprintf(msg, "*** Compute Capability < 3.0");
		return 1;
	}

	// GPU info
	sprintf(msg, "%s, %dMB, %dMP, C.C.%d.%d",
		prop.name,
		(int)(prop.totalGlobalMem / 1024 / 1024),
		prop.multiProcessorCount,
		prop.major,
		prop.minor);

	return 0;
}
