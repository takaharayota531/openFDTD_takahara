/*
cuda_memory.cu

CUDA memory utilities
*/


#include <hip/hip_runtime.h>
#include <stdlib.h>

// malloc and clear
void cuda_malloc(int gpu, int um, void **ptr, size_t size)
{
	if (gpu) {
		if (um) {
			hipMallocManaged(ptr, size);
		}
		else {
			hipMalloc(ptr, size);
		}
		hipMemset(*ptr, 0, size);
	}
	else {
		*ptr = malloc(size);
		memset(*ptr, 0, size);
	}
}

// free
void cuda_free(int gpu, void *ptr)
{
	if (gpu) {
		hipFree(ptr);
	}
	else {
		free(ptr);
	}
}

// memset
void cuda_memset(int gpu, void *ptr, int c, size_t size)
{
	if (gpu) {
		hipMemset(ptr, c, size);
	}
	else {
		memset(ptr, c, size);
	}
}

// memcpy
void cuda_memcpy(int gpu, void *dst, const void *src, size_t size, hipMemcpyKind kind)
{
	if (gpu) {
		hipMemcpy(dst, src, size, kind);
	}
	else {
		memcpy(dst, src, size);
	}
}
