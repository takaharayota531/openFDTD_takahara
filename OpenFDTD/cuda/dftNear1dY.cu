#include "hip/hip_runtime.h"
/*
dftNear1dY.cu

DFT of near Y-line
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "fieldnode.cu"

__global__
static void dftNear1dY_gpu(
	int commsize, int commrank,
	int nx, int ny, int nz, int imin, int imax,
	int k, int i,
	real_t *ex, real_t *ey, real_t *ez,
	real_t *hx, real_t *hy, real_t *hz,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz,
	d_complex_t fe, d_complex_t fh)
{
	const int j = threadIdx.x + (blockIdx.x * blockDim.x);
	if (j <= ny) {
		if (((i >= imin) && (i < imax)) || ((commrank == commsize - 1) && (i == nx))) {
			const int id = j       ;
			fieldnode(
				nx, ny, nz, imin, imax,
				i, j, k,
				ex, ey, ez,
				hx, hy, hz,
				&cex[id], &cey[id], &cez[id],
				&chx[id], &chy[id], &chz[id],
				fe, fh, &d_Param);
		}
	}
}

static void dftNear1dY_cpu(
	int commsize, int commrank,
	int nx, int ny, int nz, int imin, int imax,
	int k, int i,
	real_t *ex, real_t *ey, real_t *ez,
	real_t *hx, real_t *hy, real_t *hz,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz,
	d_complex_t fe, d_complex_t fh)
{
	for (int j = 0; j <= ny; j++) {
		if (((i >= imin) && (i < imax)) || ((commrank == commsize - 1) && (i == nx))) {
			const int id = j       ;
			fieldnode(
				nx, ny, nz, imin, imax,
				i, j, k,
				ex, ey, ez,
				hx, hy, hz,
				&cex[id], &cey[id], &cez[id],
				&chx[id], &chy[id], &chz[id],
				fe, fh, &h_Param);
		}
	}
}

void dftNear1dY(int k, int i, int64_t adr1, int64_t adr2,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dftNear1dY_gpu<<<CEIL(Ny + 1, near1dBlock), near1dBlock>>>(
			commSize, commRank,
			Nx, Ny, Nz, iMin, iMax,
			k, i,
			Ex, Ey, Ez,
			Hx, Hy, Hz,
			&d_Near1dEx[adr1], &d_Near1dEy[adr1], &d_Near1dEz[adr1],
			&d_Near1dHx[adr1], &d_Near1dHy[adr1], &d_Near1dHz[adr1],
			cEdft[adr2], cHdft[adr2]);
		if (UM) hipDeviceSynchronize();
	}
	else {
		dftNear1dY_cpu(
			commSize, commRank,
			Nx, Ny, Nz, iMin, iMax,
			k, i,
			Ex, Ey, Ez,
			Hx, Hy, Hz,
			&cex[adr1], &cey[adr1], &cez[adr1],
			&chx[adr1], &chy[adr1], &chz[adr1],
			cEdft[adr2], cHdft[adr2]);
	}
}
