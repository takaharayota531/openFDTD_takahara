#include "hip/hip_runtime.h"
/*
dftNear2dY.cu

DFT of near2d field in Y plane
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "fieldnode.cu"

__global__
static void dftNear2dY_gpu(
	int commsize, int commrank,
	int nx, int ny, int nz, int imin, int imax,
	int j,
	real_t *ex, real_t *ey, real_t *ez,
	real_t *hx, real_t *hy, real_t *hz,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz,
	d_complex_t fe, d_complex_t fh)
{
	const int i = threadIdx.y + (blockIdx.y * blockDim.y);
	const int k = threadIdx.x + (blockIdx.x * blockDim.x);
	if ((i <= nx) &&
	    (k <= nz)) {
		if (((i >= imin) && (i < imax)) || ((commrank == commsize - 1) && (i == nx))) {
			const int id = (i - imin) * (nz + 1) + k;
			fieldnode(
				nx, ny, nz, imin, imax,
				i, j, k,
				ex, ey, ez,
				hx, hy, hz,
				&cex[id], &cey[id], &cez[id],
				&chx[id], &chy[id], &chz[id],
				fe, fh, &d_Param);
		}
	}
}

static void dftNear2dY_cpu(
	int commsize, int commrank,
	int nx, int ny, int nz, int imin,int imax,
	int j,
	real_t *ex, real_t *ey, real_t *ez,
	real_t *hx, real_t *hy, real_t *hz,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz,
	d_complex_t fe, d_complex_t fh)
{
	for (int i = 0; i <= nx; i++) {
	for (int k = 0; k <= nz; k++) {
		if (((i >= imin) && (i < imax)) || ((commrank == commsize - 1) && (i == nx))) {
			const int id = (i - imin) * (nz + 1) + k;
			fieldnode(
				nx, ny, nz, imin, imax,
				i, j, k,
				ex, ey, ez,
				hx, hy, hz,
				&cex[id], &cey[id], &cez[id],
				&chx[id], &chy[id], &chz[id],
				fe, fh, &h_Param);
		}
	}
	}
}

void dftNear2dY(int j, int64_t adr1, int64_t adr2,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dim3 grid(
			CEIL(Nz + 1, near2dBlock.x),
			CEIL(Nx + 1, near2dBlock.y));
		dftNear2dY_gpu<<<grid, near2dBlock>>>(
			commSize, commRank,
			Nx, Ny, Nz, iMin, iMax,
			j,
			Ex, Ey, Ez,
			Hx, Hy, Hz,
			&d_Near2dEx[adr1], &d_Near2dEy[adr1], &d_Near2dEz[adr1],
			&d_Near2dHx[adr1], &d_Near2dHy[adr1], &d_Near2dHz[adr1],
			cEdft[adr2], cHdft[adr2]);
		if (UM) hipDeviceSynchronize();
	}
	else {
		dftNear2dY_cpu(
			commSize, commRank,
			Nx, Ny, Nz, iMin, iMax,
			j,
			Ex, Ey, Ez,
			Hx, Hy, Hz,
			&cex[adr1], &cey[adr1], &cez[adr1],
			&chx[adr1], &chy[adr1], &chz[adr1],
			cEdft[adr2], cHdft[adr2]);
	}
}
