#include "hip/hip_runtime.h"
/*
dftNear2dZ.cu

DFT of near2d field in Z plane
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "fieldnode.cu"

__global__
static void dftNear2dZ_gpu(
	int commsize, int commrank,
	int nx, int ny, int nz, int imin, int imax,
	int k,
	real_t *ex, real_t *ey, real_t *ez,
	real_t *hx, real_t *hy, real_t *hz,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz,
	d_complex_t fe, d_complex_t fh)
{
	const int i = threadIdx.y + (blockIdx.y * blockDim.y);
	const int j = threadIdx.x + (blockIdx.x * blockDim.x);
	if ((i <= nx) &&
	    (j <= ny)) {
		if (((i >= imin) && (i < imax)) || ((commrank == commsize - 1) && (i == nx))) {
			const int id = (i - imin) * (ny + 1) + j;
			fieldnode(
				nx, ny, nz, imin, imax,
				i, j, k,
				ex, ey, ez,
				hx, hy, hz,
				&cex[id], &cey[id], &cez[id],
				&chx[id], &chy[id], &chz[id],
				fe, fh, &d_Param);
		}
	}
}

static void dftNear2dZ_cpu(
	int commsize, int commrank,
	int nx, int ny, int nz, int imin,int imax,
	int k,
	real_t *ex, real_t *ey, real_t *ez,
	real_t *hx, real_t *hy, real_t *hz,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz,
	d_complex_t fe, d_complex_t fh)
{
	for (int i = 0; i <= nx; i++) {
	for (int j = 0; j <= ny; j++) {
		if (((i >= imin) && (i < imax)) || ((commrank == commsize - 1) && (i == nx))) {
			const int id = (i - imin) * (ny + 1) + j;
			fieldnode(
				nx, ny, nz, imin, imax,
				i, j, k,
				ex, ey, ez,
				hx, hy, hz,
				&cex[id], &cey[id], &cez[id],
				&chx[id], &chy[id], &chz[id],
				fe, fh, &h_Param);
		}
	}
	}
}

void dftNear2dZ(int k, int64_t adr1, int64_t adr2,
	d_complex_t *cex, d_complex_t *cey, d_complex_t *cez,
	d_complex_t *chx, d_complex_t *chy, d_complex_t *chz)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dim3 grid(
			CEIL(Ny + 1, near2dBlock.x),
			CEIL(Nx + 1, near2dBlock.y));
		dftNear2dZ_gpu<<<grid, near2dBlock>>>(
			commSize, commRank,
			Nx, Ny, Nz, iMin, iMax,
			k,
			Ex, Ey, Ez,
			Hx, Hy, Hz,
			&d_Near2dEx[adr1], &d_Near2dEy[adr1], &d_Near2dEz[adr1],
			&d_Near2dHx[adr1], &d_Near2dHy[adr1], &d_Near2dHz[adr1],
			cEdft[adr2], cHdft[adr2]);
		if (UM) hipDeviceSynchronize();
	}
	else {
		dftNear2dZ_cpu(
			commSize, commRank,
			Nx, Ny, Nz, iMin, iMax,
			k,
			Ex, Ey, Ez,
			Hx, Hy, Hz,
			&cex[adr1], &cey[adr1], &cez[adr1],
			&chx[adr1], &chy[adr1], &chz[adr1],
			cEdft[adr2], cHdft[adr2]);
	}
}
