#include "hip/hip_runtime.h"
/*
dispersionEy.cu (CUDA)

update Ey (dispersion)
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "finc_cuda.h"


__host__ __device__
static void dispersion(
	real_t e[], real_t *de, dispersion_t *me, param_t *p,
	real_t x, real_t y, real_t z, real_t t)
{
	const int64_t n = LA(p, me->i, me->j, me->k);

	real_t fi = 0;
	if (p->IPlanewave) {
		real_t dfi;
		finc_cuda(x, y, z, t, p->r0, p->ri, p->ei[1], p->ai, p->dt, &fi, &dfi);
	}

	e[n] += me->f1 * (*de);

	*de = me->f2 * (e[n] + fi)
	    + me->f3 * (*de);
}


__global__
static void dispersionEy_gpu(
	int64_t num, real_t e[], real_t de[], dispersion_t me[],
	const real_t xn[], const real_t yc[], const real_t zn[], real_t t)
	
{
	const int64_t n = threadIdx.x + (blockIdx.x * blockDim.x);
	if (n < num) {
		const real_t x = xn[me[n].i];
		const real_t y = yc[me[n].j];
		const real_t z = zn[me[n].k];
		dispersion(
			e, &de[n], &me[n], &d_Param,
			x, y, z, t);
	}
}


static void dispersionEy_cpu(
	int64_t num, real_t e[], real_t de[], dispersion_t me[],
	const real_t xn[], const real_t yc[], const real_t zn[], real_t t)
{
	for (int64_t n = 0; n < num; n++) {
		const real_t x = xn[me[n].i];
		const real_t y = yc[me[n].j];
		const real_t z = zn[me[n].k];
		dispersion(
			e, &de[n], &me[n], &h_Param,
			x, y, z, t);
	}
}


void dispersionEy(double t)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dispersionEy_gpu<<<(int)CEIL(numDispersionEy, dispersionBlock), dispersionBlock>>>(
			numDispersionEy, Ey, d_DispersionEy, d_mDispersionEy,
			d_Xn, d_Yc, d_Zn, (real_t)t);
		if (UM) hipDeviceSynchronize();
	}
	else {
		dispersionEy_cpu(
			numDispersionEy, Ey, DispersionEy, mDispersionEy,
			h_Xn, h_Yc, h_Zn, (real_t)t);
	}
}
