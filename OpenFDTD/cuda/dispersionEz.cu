#include "hip/hip_runtime.h"
/*
dispersionEz.cu (CUDA)

update Ez (dispersion)
*/

#include "ofd.h"
#include "ofd_cuda.h"
#include "finc_cuda.h"


__host__ __device__
static void dispersion(
	real_t e[], real_t *de, dispersion_t *me, param_t *p,
	real_t x, real_t y, real_t z, real_t t)
{
	const int64_t n = LA(p, me->i, me->j, me->k);

	real_t fi = 0;
	if (p->IPlanewave) {
		real_t dfi;
		finc_cuda(x, y, z, t, p->r0, p->ri, p->ei[2], p->ai, p->dt, &fi, &dfi);
	}

	e[n] += me->f1 * (*de);

	*de = me->f2 * (e[n] + fi)
	    + me->f3 * (*de);
}


__global__
static void dispersionEz_gpu(
	int64_t num, real_t e[], real_t de[], dispersion_t me[],
	const real_t xn[], const real_t yn[], const real_t zc[], real_t t)
	
{
	const int64_t n = threadIdx.x + (blockIdx.x * blockDim.x);
	if (n < num) {
		const real_t x = xn[me[n].i];
		const real_t y = yn[me[n].j];
		const real_t z = zc[me[n].k];
		dispersion(
			e, &de[n], &me[n], &d_Param,
			x, y, z, t);
	}
}


static void dispersionEz_cpu(
	int64_t num, real_t e[], real_t de[], dispersion_t me[],
	const real_t xn[], const real_t yn[], const real_t zc[], real_t t)
{
	for (int64_t n = 0; n < num; n++) {
		const real_t x = xn[me[n].i];
		const real_t y = yn[me[n].j];
		const real_t z = zc[me[n].k];
		dispersion(
			e, &de[n], &me[n], &h_Param,
			x, y, z, t);
	}
}


void dispersionEz(double t)
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dispersionEz_gpu<<<(int)CEIL(numDispersionEz, dispersionBlock), dispersionBlock>>>(
			numDispersionEz, Ez, d_DispersionEz, d_mDispersionEz,
			d_Xn, d_Yn, d_Zc, (real_t)t);
		if (UM) hipDeviceSynchronize();
	}
	else {
		dispersionEz_cpu(
			numDispersionEz, Ez, DispersionEz, mDispersionEz,
			h_Xn, h_Yn, h_Zc, (real_t)t);
	}
}
