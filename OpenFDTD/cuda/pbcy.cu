#include "hip/hip_runtime.h"
/*
pbcy.cu (CUDA)

PBC on +/- Y boundary
*/

#include "ofd.h"
#include "ofd_cuda.h"


__host__ __device__
static void _pbcyhz(int k, int i, float *hz, param_t *p)
{
	hz[LA(p, i,    -1, k)] = hz[LA(p, i, p->Ny - 1, k)];
	hz[LA(p, i, p->Ny, k)] = hz[LA(p, i,         0, k)];
}


__host__ __device__
static void _pbcyhx(int k, int i, float *hx, param_t *p)
{
	hx[LA(p, i,    -1, k)] = hx[LA(p, i, p->Ny - 1, k)];
	hx[LA(p, i, p->Ny, k)] = hx[LA(p, i,         0, k)];
}


__global__
static void pbcyhz_gpu(float *hz)
{
	int k = d_Param.kMin - 0 + (blockDim.x * blockIdx.x) + threadIdx.x;
	int i = d_Param.iMin - 1 + (blockDim.y * blockIdx.y) + threadIdx.y;

	if (k <= d_Param.kMax) {
	if (i <= d_Param.iMax) {
		_pbcyhz(k, i, hz, &d_Param);
	}
	}
}


__global__
static void pbcyhx_gpu(float *hx)
{
	int k = d_Param.kMin - 1 + (blockDim.x * blockIdx.x) + threadIdx.x;
	int i = d_Param.iMin - 0 + (blockDim.y * blockIdx.y) + threadIdx.y;

	if (k <= d_Param.kMax) {
	if (i <= d_Param.iMax) {
		_pbcyhx(k, i, hx, &d_Param);
	}
	}
}


static void pbcyhz_cpu(float *hz)
{
	for (int k = h_Param.kMin - 0; k <= h_Param.kMax; k++) {
	for (int i = h_Param.iMin - 1; i <= h_Param.iMax; i++) {
		_pbcyhz(k, i, hz, &h_Param);
	}
	}
}


static void pbcyhx_cpu(float *hx)
{
	for (int k = h_Param.kMin - 1; k <= h_Param.kMax; k++) {
	for (int i = h_Param.iMin - 0; i <= h_Param.iMax; i++) {
		_pbcyhx(k, i, hx, &h_Param);
	}
	}
}


void pbcy()
{
	if (GPU) {
		hipMemcpyToSymbol(HIP_SYMBOL(d_Param), &h_Param, sizeof(param_t));
		dim3 block(pbcBlock, pbcBlock);
		dim3 grid_hz(CEIL(kMax - kMin + 1, block.x),
		             CEIL(iMax - iMin + 2, block.y));
		dim3 grid_hx(CEIL(kMax - kMin + 2, block.x),
		             CEIL(iMax - iMin + 1, block.y));
		pbcyhz_gpu<<<grid_hz, block>>>(Hz);
		pbcyhx_gpu<<<grid_hx, block>>>(Hx);
		if (UM) hipDeviceSynchronize();
	}
	else {
		pbcyhz_cpu(Hz);
		pbcyhx_cpu(Hx);
	}
}
