#include "hip/hip_runtime.h"
/*
solve.cu (CUDA + MPI)
*/

#ifdef _MPI
#include <mpi.h>
#endif

#include "ofd.h"
#include "ofd_mpi.h"
#include "ofd_cuda.h"
#include "ofd_prototype.h"


void solve(int io, int mem, FILE *fp)
{
	double fmax[] = {0, 0};
	char   str[BUFSIZ];
	int    converged = 0;

	// setup host memory
	setup_host();

	// setup (GPU)
	if (GPU) {
		setup_gpu();
	}

	// initial field
	initfield();

	// time step iteration
	int itime;
	double t = 0;
	for (itime = 0; itime <= Solver.maxiter; itime++) {

		// update H
		t += 0.5 * Dt;
		updateHx(t);
		updateHy(t);
		updateHz(t);

		// ABC H
		if      (iABC == 0) {
			murHx();
			murHy();
			murHz();
		}
		else if (iABC == 1) {
			pmlHx();
			pmlHy();
			pmlHz();
		}

		// PBC H
		if (PBCx) {
			if (commSize > 1) {
				comm_cuda_pbcx();
			}
			else {
				pbcx();
			}
		}
		if (PBCy) {
			pbcy();
		}
		if (PBCz) {
			pbcz();
		}

		// share boundary H (MPI)
		if (commSize > 1) {
			comm_cuda_boundary();
		}

		// update E
		t += 0.5 * Dt;
		updateEx(t);
		updateEy(t);
		updateEz(t);

		// dispersion E
		if (numDispersionEx) {
			dispersionEx(t);
		}
		if (numDispersionEy) {
			dispersionEy(t);
		}
		if (numDispersionEz) {
			dispersionEz(t);
		}

		// ABC E
		if      (iABC == 1) {
			pmlEx();
			pmlEy();
			pmlEz();
		}

		// feed
		if (NFeed) {
			efeed(itime);
		}

		// inductor
		if (NInductor) {
			eload();
		}

		// point
		if (NPoint) {
			vpoint(itime);
		}

		// DFT
		if      (runMode == 0) {
			dftNear1d(itime, l_LNear1d, l_Near1dEx, l_Near1dEy, l_Near1dEz, l_Near1dHx, l_Near1dHy, l_Near1dHz);
			dftNear2d(itime, l_LNear2d, l_Near2dEx, l_Near2dEy, l_Near2dEz, l_Near2dHx, l_Near2dHy, l_Near2dHz);
		}
		else if (runMode == 1) {
			dftNear3d(itime);
		}

		// average and convergence
		if ((itime % Solver.nout == 0) || (itime == Solver.maxiter)) {
			// average
			double fsum[2];
			average(fsum);

			// allreduce average (MPI)
			if (commSize > 1) {
				comm_average(fsum);
			}

			// average
			if (commRank == 0) {
				Eiter[Niter] = fsum[0];
				Hiter[Niter] = fsum[1];
				Niter++;
			}

			// monitor
			if (io) {
				sprintf(str, "%7d %.6f %.6f", itime, fsum[0], fsum[1]);
				fprintf(fp,     "%s\n", str);
				fprintf(stdout, "%s\n", str);
				fflush(fp);
				fflush(stdout);
			}

			// check convergence
			fmax[0] = MAX(fmax[0], fsum[0]);
			fmax[1] = MAX(fmax[1], fsum[1]);
			if ((fsum[0] < fmax[0] * Solver.converg) &&
			    (fsum[1] < fmax[1] * Solver.converg)) {
				converged = 1;
				break;
			}
		}
	}

	// result
	if (io) {
		sprintf(str, "    --- %s ---", (converged ? "converged" : "max steps"));
		fprintf(fp,     "%s\n", str);
		fprintf(stdout, "%s\n", str);
		fflush(fp);
		fflush(stdout);
	}

	// time steps
	Ntime = itime + converged;

	// copy device to host
	if (GPU) {
		if (NFeed > 0) {
			cuda_memcpy(GPU, VFeed, d_VFeed, Feed_size, hipMemcpyDeviceToHost);
			cuda_memcpy(GPU, IFeed, d_IFeed, Feed_size, hipMemcpyDeviceToHost);
		}

		if (NPoint > 0) {
			cuda_memcpy(GPU, VPoint, d_VPoint, Point_size, hipMemcpyDeviceToHost);
		}

		if      (runMode == 0) {
			if ((NNear1d > 0) && (NFreq2 > 0)) {
				cuda_memcpy(GPU, l_Near1dEx, d_Near1dEx, Near1d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near1dEy, d_Near1dEy, Near1d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near1dEz, d_Near1dEz, Near1d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near1dHx, d_Near1dHx, Near1d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near1dHy, d_Near1dHy, Near1d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near1dHz, d_Near1dHz, Near1d_size, hipMemcpyDeviceToHost);
			}

			if ((NNear2d > 0) && (NFreq2 > 0)) {
				cuda_memcpy(GPU, l_Near2dEx, d_Near2dEx, Near2d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near2dEy, d_Near2dEy, Near2d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near2dEz, d_Near2dEz, Near2d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near2dHx, d_Near2dHx, Near2d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near2dHy, d_Near2dHy, Near2d_size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, l_Near2dHz, d_Near2dHz, Near2d_size, hipMemcpyDeviceToHost);
			}
		}
		else if (runMode == 1) {
			if ((NN > 0) && (NFreq2 > 0)) {
				size_t size = NN * NFreq2 * sizeof(real_t);
				cuda_memcpy(GPU, Ex_r, d_Ex_r, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Ey_r, d_Ey_r, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Ez_r, d_Ez_r, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Hx_r, d_Hx_r, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Hy_r, d_Hy_r, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Hz_r, d_Hz_r, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Ex_i, d_Ex_i, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Ey_i, d_Ey_i, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Ez_i, d_Ez_i, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Hx_i, d_Hx_i, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Hy_i, d_Hy_i, size, hipMemcpyDeviceToHost);
				cuda_memcpy(GPU, Hz_i, d_Hz_i, size, hipMemcpyDeviceToHost);
			}
		}
	}

	// free
	if (mem) {
		memfree2_gpu();
	}

	// MPI : send to root
	if (commSize > 1) {
		// feed waveform
		if (NFeed > 0) {
			comm_feed();
		}

		// point waveform
		if (NPoint > 0) {
			comm_point();
		}

		// near field
		if      (runMode == 0) {
			// near1d
			if ((NNear1d > 0) && (NFreq2 > 0)) {
				comm_near1d();
			}

			// near2d
			if ((NNear2d > 0) && (NFreq2 > 0)) {
				comm_near2d();
			}
		}
		else if (runMode == 1) {
			// near3d
			if (NFreq2 > 0) {
				comm_near3d();
			}
		}
	}

	// non-MPI: copy pointer local to global
	else {
		if      (runMode == 0) {
			LNear1d  = l_LNear1d;
			Near1dEx = l_Near1dEx;
			Near1dEy = l_Near1dEy;
			Near1dEz = l_Near1dEz;
			Near1dHx = l_Near1dHx;
			Near1dHy = l_Near1dHy;
			Near1dHz = l_Near1dHz;

			LNear2d  = l_LNear2d;
			Near2dEx = l_Near2dEx;
			Near2dEy = l_Near2dEy;
			Near2dEz = l_Near2dEz;
			Near2dHx = l_Near2dHx;
			Near2dHy = l_Near2dHy;
			Near2dHz = l_Near2dHz;
		}
		else if (runMode == 1) {
			for (int ic = 0; ic < 6; ic++) {
				if (mem) calcNear3d(ic, 0);
				calcNear3d(ic, 1);
				if (mem) calcNear3d(ic, 2);
			}
		}
	}
}
